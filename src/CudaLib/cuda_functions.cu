/**
 * @file   cuda_functions.cu
 * @brief  Basic functions of CUDA.
 * @date   Nov 4, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include <stdio.h>

float* cuda_alloc_float(int size)
{
	float *d;

	hipError_t error = hipMalloc((void **) &d, size * sizeof(float));

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMalloc failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return d;
}

int* cuda_alloc_int(int size)
{
	int *d;

	hipError_t error = hipMalloc((void **) &d, size * sizeof(int));

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMalloc failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return d;
}

void cuda_fill_zero(float* d, int size)
{
	hipError_t error = hipMemset(d, 0, size * sizeof(float));

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMemset failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void cuda_free(float* d)
{
	hipError_t error = hipFree(d);

	if (error != hipSuccess)
	{
		fprintf(stderr, "hipFree failed (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void cuda_free(int* d)
{
	hipError_t error = hipFree(d);

	if (error != hipSuccess)
	{
		fprintf(stderr, "hipFree failed (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void cuda_copyHostToDevice_float(float *dest, float *source, int size)
{
	hipError_t error = hipMemcpy(dest, source, size * sizeof(float), hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMemcpy HostToDevice float failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void cuda_copyHostToDevice_int(int *dest, int *source, int size)
{
	hipError_t error = hipMemcpy(dest, source, size * sizeof(int), hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMemcpy HostToDevice int failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void cuda_copyDeviceToHost_float(float *dest, float *source, int size)
{
	hipError_t error = hipMemcpy(dest, source, size * sizeof(float), hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy DeviceToHost float failed (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void cuda_copyDeviceToHost_int(int *dest, int *source, int size)
{
	hipError_t error = hipMemcpy(dest, source, size * sizeof(int), hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy DeviceToHost int failed (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

int cuda_getNumberOfGPUs()
{
    int GPU_N;
    hipError_t error = hipGetDeviceCount(&GPU_N);

    if (error != hipSuccess)
    {
        fprintf(stderr, "cuda_numberOfGPUs failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return GPU_N;
}

void cuda_setDevice(int number)
{
    hipError_t error = hipSetDevice(number);

    if (error != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}
