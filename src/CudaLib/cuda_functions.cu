/**
 * @file   cuda_functions.cu
 * @brief  Basic functions of CUDA.
 * @date   Nov 4, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include <stdio.h>

float* cuda_alloc_float(int size)
{
	float *d;

	hipError_t error = hipMalloc((void **) &d, size * sizeof(float));

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMalloc failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return d;
}
void cuda_fill_zero(float* d, int size)
{
	hipError_t error = hipMemset(d, 0, size * sizeof(float));

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMemset failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void cuda_free(float* d)
{
	hipError_t error = hipFree(d);

	if (error != hipSuccess)
	{
		fprintf(stderr, "hipFree failed (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void cuda_copyHostToDevice_float(float *h, float *d, int size)
{
	hipError_t error = hipMemcpy(d, h, size * sizeof(float), hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMemcpy failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void cuda_copyDeviceToHost_float(float *d, float *h, int size)
{
	hipError_t error = hipMemcpy(h, d, size * sizeof(float), hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}
