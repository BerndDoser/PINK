/**
 * @file   cuda_functions.cu
 * @brief  Basic functions of CUDA.
 * @date   Nov 4, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include <stdio.h>

float* cuda_alloc_float(int size)
{
	float *d;

	hipError_t error = hipMalloc((void **) &d, size * sizeof(float));

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMalloc failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return d;
}

int* cuda_alloc_int(int size)
{
	int *d;

	hipError_t error = hipMalloc((void **) &d, size * sizeof(int));

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMalloc failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return d;
}

void cuda_fill_zero(float* d, int size)
{
	hipError_t error = hipMemset(d, 0, size * sizeof(float));

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMemset failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void cuda_free(float* d)
{
	hipError_t error = hipFree(d);

	if (error != hipSuccess)
	{
		fprintf(stderr, "hipFree failed (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void cuda_free(int* d)
{
	hipError_t error = hipFree(d);

	if (error != hipSuccess)
	{
		fprintf(stderr, "hipFree failed (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void cuda_copyHostToDevice_float(float *h, float *d, int size)
{
	hipError_t error = hipMemcpy(d, h, size * sizeof(float), hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMemcpy HostToDevice float failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void cuda_copyHostToDevice_int(int *h, int *d, int size)
{
	hipError_t error = hipMemcpy(d, h, size * sizeof(int), hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
		fprintf(stderr, "hipMemcpy HostToDevice int failed (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void cuda_copyDeviceToHost_float(float *d, float *h, int size)
{
	hipError_t error = hipMemcpy(h, d, size * sizeof(float), hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy DeviceToHost float failed (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void cuda_copyDeviceToHost_int(int *d, int *h, int size)
{
	hipError_t error = hipMemcpy(h, d, size * sizeof(int), hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy DeviceToHost int failed (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}
