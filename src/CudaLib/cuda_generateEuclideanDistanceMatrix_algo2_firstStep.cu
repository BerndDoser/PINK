#include "hip/hip_runtime.h"
/**
 * @file   CudaLib/cuda_generateEuclideanDistanceMatrix_algo2_firstStep.cu
 * @date   Oct 30, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include "hipblas.h"
#include <stdio.h>

#define BLOCK_SIZE 32

/**
 * CUDA Kernel Device code
 *
 * Computes the euclidean distance of two arrays.
 */
template <unsigned int block_size>
__global__ void
euclidean_distance_kernel(float *som, float *rotatedImages, float *firstStep, int image_size)
{
	int tid = threadIdx.x;
    int i = threadIdx.x;
    float diff;

	__shared__ float firstStep_local[block_size];
	firstStep_local[tid] = 0.0f;

	__syncthreads();

    while(i < image_size)
    {
    	diff = som[i + blockIdx.y * image_size] - rotatedImages[i + blockIdx.x * image_size];
    	firstStep_local[tid] += diff * diff;
    	i += block_size;
    	__syncthreads();
    }

	// Parallel reduction
    if (block_size >= 512) { if (tid < 256) { firstStep_local[tid] += firstStep_local[tid + 256]; } __syncthreads(); }
    if (block_size >= 256) { if (tid < 128) { firstStep_local[tid] += firstStep_local[tid + 128]; } __syncthreads(); }
    if (block_size >= 128) { if (tid <  64) { firstStep_local[tid] += firstStep_local[tid +  64]; } __syncthreads(); }

	if (tid < 32)
	{
		if (block_size >= 64) { firstStep_local[tid] += firstStep_local[tid + 32]; __syncthreads(); }
		if (block_size >= 32) { firstStep_local[tid] += firstStep_local[tid + 16]; __syncthreads(); }
		if (block_size >= 16) { firstStep_local[tid] += firstStep_local[tid +  8]; __syncthreads(); }
		if (block_size >=  8) { firstStep_local[tid] += firstStep_local[tid +  4]; __syncthreads(); }
		if (block_size >=  4) { firstStep_local[tid] += firstStep_local[tid +  2]; __syncthreads(); }
		if (block_size >=  2) { firstStep_local[tid] += firstStep_local[tid +  1]; __syncthreads(); }
	}

	// Copy accumulated local value to global array firstStep
	if (tid == 0) atomicExch(firstStep + blockIdx.x + blockIdx.y * gridDim.x, firstStep_local[tid]);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
void cuda_generateEuclideanDistanceMatrix_algo2_firstStep(float *d_som, float *d_rotatedImages,
    float* d_firstStep, int som_size, int num_rot, int image_size)
{
    // Setup execution parameters
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(num_rot, som_size);

    // Start kernel
    euclidean_distance_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_firstStep, image_size);

    hipError_t error = hipGetLastError();

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch CUDA kernel cuda_generateEuclideanDistanceMatrix_algo2_firstStep (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}
