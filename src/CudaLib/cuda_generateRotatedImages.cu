#include "hip/hip_runtime.h"
/**
 * @file   cuda_generateEuclideanDistanceMatrix_algo2_firstStep.cu
 * @date   Oct 30, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include "hipblas.h"
#include <stdio.h>

#define BLOCK_SIZE 32

texture<float, 2, hipReadModeElementType> image_texture;

/**
 * CUDA Kernel Device code
 *
 * Computes multiple rotations of an image. cosine and sin
 */
template <unsigned int block_size>
__global__ void
cuda_generateRotatedImages_kernel(float *rotatedImages, float *image, int neuron_size,
    int neuron_dim, int image_dim, float* cosAlpha, float *sinAlpha)
{
	int x2 = blockIdx.x * blockDim.x + threadIdx.x;
	int y2 = blockIdx.y * blockDim.y + threadIdx.y;

	if (x2 >= neuron_dim or y2 >= neuron_dim) return;

	int x0 = image_dim * 0.5;
	int y0 = image_dim * 0.5;
	int margin = (image_dim - neuron_dim) * 0.5;
	int x0margin = x0 - margin;
	int y0margin = y0 - margin;

	float cosAlpha_local = cosAlpha[blockIdx.z];
	float sinAlpha_local = sinAlpha[blockIdx.z];

	int x1 = (x2-x0margin)*cosAlpha_local + (y2-y0margin)*sinAlpha_local + x0;
	int y1 = (y2-y0margin)*cosAlpha_local - (x2-x0margin)*sinAlpha_local + y0;

	float *pCurRot = rotatedImages + blockIdx.z * neuron_size;

    //pCurRot[x*neuron_dim + y] = tex2D(image_texture, tx+0.5f, ty+0.5f);

    if (x1 >= 0 and x1 < image_dim and y1 >= 0 and y1 < image_dim) {
    	atomicAdd(pCurRot + x2*neuron_dim + y2, image[x1*image_dim + y1]);
    } else {
    	atomicAdd(pCurRot + x2*neuron_dim + y2, 0.0f);
    }
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
void cuda_generateRotatedImages(float* d_rotatedImages, float* d_image, int num_rot, int image_dim, int neuron_dim,
    bool flip, float *d_cosAlpha, float *d_sinAlpha)
{
	int neuron_size = neuron_dim * neuron_dim;

	cuda_fill_zero(d_rotatedImages, num_rot * neuron_size);

	//hipBindTexture(0, image_texture, d_image, image_size * sizeof(float));

	// Copy original image on first position
	//crop(image_dim, image_dim, neuron_dim, neuron_dim, image, rotatedImages);

    // Setup execution parameters
	int gridSize = ceil((float)neuron_dim/BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(gridSize, gridSize, num_rot);

    // Start kernel
    cuda_generateRotatedImages_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_rotatedImages, d_image,
        neuron_size, neuron_dim, image_dim, d_cosAlpha, d_sinAlpha);

    hipError_t error = hipGetLastError();

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch CUDA kernel cuda_generateRotatedImages (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}
