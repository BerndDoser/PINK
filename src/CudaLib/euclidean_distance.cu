
#include <hip/hip_runtime.h>
/**
 * @file   CudaTest/mixed_precision.cpp
 * @date   Apr 16, 2018
 * @author Bernd Doser <bernd.doser@h-its.org>
 */

__global__
void cuda_euclidean_distance(float *a1, float *a2, size_t size)
{

}

void euclidean_distance(float *a1, float *a2, size_t size)
{
	int8_t *ia;
	hipMalloc((void **) &ia, size * sizeof(int8_t));

	//cuda_convert_float_to_int8<<<1, 1>>>(ia, a1, size);

    cuda_euclidean_distance<<<1, 1>>>(a1, a2, size);
}
