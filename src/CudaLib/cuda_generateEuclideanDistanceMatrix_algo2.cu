#include "hip/hip_runtime.h"
/**
 * @file   cuda_calculateEuclideanDistanceWithoutSquareRoot.cu
 * @date   Oct 30, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include "hipblas.h"
#include <stdio.h>

#define BLOCK_SIZE 32

/**
 * CUDA Kernel Device code
 *
 * Computes the euclidean distance of two arrays.
 */
template <unsigned int block_size>
__global__ void
euclidean_distance_kernel(float *som, float *rotatedImages, float *firstStep, int image_size)
{
	int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float diff;

	__shared__ float firstStep_local[block_size];
	firstStep_local[tid] = 0.0f;

    while(i < image_size)
    {
    	diff = som[i + blockIdx.y * image_size] - rotatedImages[i + blockIdx.z * image_size];
    	firstStep_local[tid] += diff * diff;
    	i += block_size;
    	__syncthreads();
    }

	// Parallel reduction
	if (tid < 32)
	{
		if (block_size >= 64) { firstStep_local[tid] += firstStep_local[tid + 32]; __syncthreads(); }
		if (block_size >= 32) { firstStep_local[tid] += firstStep_local[tid + 16]; __syncthreads(); }
		if (block_size >= 16) { firstStep_local[tid] += firstStep_local[tid +  8]; __syncthreads(); }
		if (block_size >=  8) { firstStep_local[tid] += firstStep_local[tid +  4]; __syncthreads(); }
		if (block_size >=  4) { firstStep_local[tid] += firstStep_local[tid +  2]; __syncthreads(); }
		if (block_size >=  2) { firstStep_local[tid] += firstStep_local[tid +  1]; __syncthreads(); }
	}

	if (tid == 0) firstStep[blockIdx.y*blockDim.z + blockIdx.z] += firstStep_local[0];
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
void cuda_generateEuclideanDistanceMatrix_algo2(float *d_euclideanDistanceMatrix, int *d_bestRotationMatrix, int som_dim, float* d_som,
    int image_dim, int num_rot, float* d_rotatedImages)
{
	unsigned int image_size = image_dim * image_dim;
	unsigned int som_size = som_dim * som_dim;
	//unsigned int red_size = ceil((float)image_size/BLOCK_SIZE);
	unsigned int red_size = 1;

    // Setup execution parameters
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(red_size, som_size, num_rot);

    float *d_firstStep = cuda_alloc_float(som_size * num_rot);
    cuda_fill_zero(d_firstStep, som_size * num_rot);

    // Start kernel
    //printf("Starting CUDA Kernel with (%i,%i,%i) blocks and (%i,%i,%i) threads ...\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);
    euclidean_distance_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_firstStep, image_size);

    hipError_t error = hipGetLastError();

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch CUDA kernel cuda_generateEuclideanDistanceMatrix_algo2 (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

	float *firstStep = new float[som_size * num_rot];
	cuda_copyDeviceToHost_float(firstStep, d_firstStep, som_size * num_rot);

	for (int i=0; i < som_size; ++i)
		printf("gpu eucl %i: %f\n", i, firstStep[i*num_rot]);

    cuda_generateEuclideanDistanceMatrix_algo2_secondStep(d_euclideanDistanceMatrix, d_bestRotationMatrix,
        d_firstStep, som_size, num_rot);

    cuda_free(d_firstStep);
}
