#include "hip/hip_runtime.h"
/**
 * @file   cuda_calculateEuclideanDistanceWithoutSquareRoot.cu
 * @date   Oct 30, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include <stdio.h>

#define BLOCK_SIZE 32

/**
 * CUDA Kernel Device code
 *
 * Computes the euclidean norm of array a and b.
 */
__global__ void
kernel(float *a, float *b, float *c, int length)
{
	int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ float a_local[BLOCK_SIZE];
    __shared__ float b_local[BLOCK_SIZE];
    __shared__ float c_local[BLOCK_SIZE];

    a_local[tid] = (i < length) ? a[i] : 0.0;
    b_local[tid] = (i < length) ? b[i] : 0.0;
    c_local[tid] = 0.0;
    __syncthreads();

    float tmp = a_local[i] - b_local[i];
    c_local[tid] += tmp * tmp;

    for (int s=1; s < blockDim.x; s *= 2) {
    	if (tid % (2*s) == 0) {
    	    c_local[tid] += c_local[tid + s];
    	}
    	__syncthreads();
    }

    if (tid == 0)
        for (int j=0; j < BLOCK_SIZE; j++)
            *c += c_local[j];
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
float cuda_calculateEuclideanDistanceWithoutSquareRoot(float *a, float *b, int length)
{
	float c = 0.0;
    unsigned int sizeInBytes = length * sizeof(float);

    // Allocate device memory
    float *d_a, *d_b, *d_c;

    hipError_t error;

    error = hipMalloc((void **) &d_a, sizeInBytes);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_a returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_b, sizeInBytes);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_b returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_c, sizeof(float));

    if (error != hipSuccess)
    {
        printf("hipMalloc d_c returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_a, a, sizeInBytes, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy d_a returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_b, b, sizeInBytes, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy d_b returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_c, &c, sizeof(float), hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy d_b returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }


    // Setup execution parameters
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(length/BLOCK_SIZE);

    printf("Starting CUDA Kernel with (%i,%i,%i) blocks and (%i,%i,%i) threads ...\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);
    kernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, length);

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Copy the device result vector in device memory to the host result vector in host memory.
    error = hipMemcpy(&c, d_c, sizeof(float), hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to copy d_c to host (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    error = hipFree(d_a);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_a (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipFree(d_b);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_b (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipFree(d_c);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_c (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return c;
}
