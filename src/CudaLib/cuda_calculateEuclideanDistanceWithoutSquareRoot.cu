#include "hip/hip_runtime.h"
/**
 * @file   cuda_calculateEuclideanDistanceWithoutSquareRoot.cu
 * @date   Oct 30, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include <stdio.h>

#define BLOCK_SIZE 32

/**
 * CUDA Kernel Device code
 *
 * Computes the euclidean norm of array a and b.
 */
__global__ void
kernel(float *a, float *b, float *c, int length)
{
	int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ float a_local[BLOCK_SIZE];
    __shared__ float b_local[BLOCK_SIZE];

    a_local[tid] = (i < length) ? a[i] : 0.0;
    b_local[tid] = (i < length) ? b[i] : 0.0;
    __syncthreads();

    float tmp;
    tmp = a_local[i] - b_local[i];
    *c += tmp * tmp;
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
float cuda_calculateEuclideanDistanceWithoutSquareRoot(float *a, float *b, int length)
{
	float c = 0.0;
    unsigned int sizeInBytes = length * sizeof(float);

    // Allocate device memory
    float *d_a, *d_b, *d_c;

    hipError_t error;

    error = hipMalloc((void **) &d_a, sizeInBytes);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_a returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_b, sizeInBytes);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_b returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_c, sizeof(float));

    if (error != hipSuccess)
    {
        printf("hipMalloc d_c returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_a, a, sizeInBytes, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy d_a returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_b, b, sizeInBytes, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy d_b returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_c, &c, sizeof(float), hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy d_b returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }


    // Setup execution parameters
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(length/BLOCK_SIZE);

    printf("Starting CUDA Kernel with (%i,%i,%i) blocks and (%i,%i,%i) threads ...\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);

    kernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, length);

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Free device global memory
    error = hipFree(d_a);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_a (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipFree(d_b);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_b (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return c;
}
