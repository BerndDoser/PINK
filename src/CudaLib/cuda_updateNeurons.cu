#include "hip/hip_runtime.h"
/**
 * @file   CudaLib/cuda_updateNeurons.cu
 * @date   Nov 13, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include "updateNeurons_kernel.h"
#include <float.h>
#include <stdio.h>

#define BLOCK_SIZE 32

/**
 * CUDA Kernel Device code
 *
 * Find the position where the euclidean distance is minimal between image and neuron.
 */
__global__ void
findBestMatchingNeuron_kernel(float *euclideanDistanceMatrix, int *bestMatch, int som_size)
{
    *bestMatch = 0;
    float minDistance = euclideanDistanceMatrix[0];
    for (int i = 1; i < som_size; ++i) {
        if (euclideanDistanceMatrix[i] < minDistance) {
            minDistance = euclideanDistanceMatrix[i];
            *bestMatch = i;
        }
    }
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
void cuda_updateNeurons(float *d_som, float *d_rotatedImages, int *d_bestRotationMatrix, float *d_euclideanDistanceMatrix,
    int* d_bestMatch, int som_width, int som_height, int som_depth, int som_size, int neuron_size, int num_rot,
    Function function, Layout layout, float sigma, float damping, float maxUpdateDistance, bool usePBC, int dimensionality)
{
    {
        // Start kernel
        findBestMatchingNeuron_kernel<<<1,1>>>(d_euclideanDistanceMatrix, d_bestMatch, som_size);

        hipError_t error = hipGetLastError();

        if (error != hipSuccess)
        {
            fprintf(stderr, "Failed to launch CUDA kernel findBestMatchingNeuron_kernel (error code %s)!\n", hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }
    }
    {
        // Setup execution parameters
        int gridSize = ceil((float)neuron_size/BLOCK_SIZE);
        dim3 dimBlock(BLOCK_SIZE);
        dim3 dimGrid(gridSize, som_size);

        // Start kernel
        if (function == GAUSSIAN) {
            if (layout == QUADRATIC) {
                if (usePBC) {
                    if (dimensionality == 1) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, GaussianFunctor(sigma), CartesianDistanceFunctor<1, true>(som_width),
                            damping, maxUpdateDistance);
                    } else if (dimensionality == 2) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, GaussianFunctor(sigma), CartesianDistanceFunctor<2, true>(som_width, som_height),
                            damping, maxUpdateDistance);
                    } else if (dimensionality == 3) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, GaussianFunctor(sigma), CartesianDistanceFunctor<3, true>(som_width, som_height, som_depth),
                            damping, maxUpdateDistance);
                    }
                } else {
                    if (dimensionality == 1) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, GaussianFunctor(sigma), CartesianDistanceFunctor<1>(som_width),
                            damping, maxUpdateDistance);
                    } else if (dimensionality == 2) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, GaussianFunctor(sigma), CartesianDistanceFunctor<2>(som_width, som_height),
                            damping, maxUpdateDistance);
                    } else if (dimensionality == 3) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, GaussianFunctor(sigma), CartesianDistanceFunctor<3>(som_width, som_height, som_depth),
                            damping, maxUpdateDistance);
                    }
                }
            } else if (layout == HEXAGONAL) {
                updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                    d_bestMatch, neuron_size, GaussianFunctor(sigma), HexagonalDistanceFunctor(som_width),
                    damping, maxUpdateDistance);
            }
        } else if (function == MEXICANHAT) {
            if (layout == QUADRATIC) {
                if (usePBC) {
                    if (dimensionality == 1) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, MexicanHatFunctor(sigma), CartesianDistanceFunctor<1, true>(som_width),
                            damping, maxUpdateDistance);
                    } else if (dimensionality == 2) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, MexicanHatFunctor(sigma), CartesianDistanceFunctor<2, true>(som_width, som_height),
                            damping, maxUpdateDistance);
                    } else if (dimensionality == 3) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, MexicanHatFunctor(sigma), CartesianDistanceFunctor<3, true>(som_width, som_height, som_depth),
                            damping, maxUpdateDistance);
                    }
                } else {
                    if (dimensionality == 1) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, MexicanHatFunctor(sigma), CartesianDistanceFunctor<1>(som_width),
                            damping, maxUpdateDistance);
                    } else if (dimensionality == 2) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, MexicanHatFunctor(sigma), CartesianDistanceFunctor<2>(som_width, som_height),
                            damping, maxUpdateDistance);
                    } else if (dimensionality == 3) {
                        updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                            d_bestMatch, neuron_size, MexicanHatFunctor(sigma), CartesianDistanceFunctor<3>(som_width, som_height, som_depth),
                            damping, maxUpdateDistance);
                    }
                }
            } else if (layout == HEXAGONAL) {
                updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
                    d_bestMatch, neuron_size, MexicanHatFunctor(sigma), HexagonalDistanceFunctor(som_width),
                    damping, maxUpdateDistance);
            }
        }

        hipError_t error = hipGetLastError();

        if (error != hipSuccess)
        {
            fprintf(stderr, "Failed to launch CUDA kernel updateNeurons_kernel (error code %s)!\n", hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }
    }
}
