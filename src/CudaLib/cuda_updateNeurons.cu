#include "hip/hip_runtime.h"
/**
 * @file   CudaLib/cuda_updateNeurons.cu
 * @date   Nov 13, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include "updateNeurons_kernel.h"
#include "hipblas.h"
#include <float.h>
#include <stdio.h>

#define BLOCK_SIZE 32

/**
 * CUDA Kernel Device code
 *
 * Find the position where the euclidean distance is minimal between image and neuron.
 */
__global__ void
findBestMatchingNeuron_kernel(float *euclideanDistanceMatrix, int *bestMatch, int som_dim)
{
    float minDistance = FLT_MAX;

    for (int i = 0, ij = 0; i < som_dim; ++i) {
        for (int j = 0; j < som_dim; ++j, ++ij) {
			if (euclideanDistanceMatrix[ij] < minDistance) {
				minDistance = euclideanDistanceMatrix[ij];
				bestMatch[0] = i;
				bestMatch[1] = j;
			}
		}
    }
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
void cuda_updateNeurons(float *d_som, float *d_rotatedImages, int *d_bestRotationMatrix, float *d_euclideanDistanceMatrix,
    int* d_bestMatch, int som_dim, int neuron_dim, int num_rot, int numberOfChannels)
{
    {
    	// Start kernel
        findBestMatchingNeuron_kernel<<<1,1>>>(d_euclideanDistanceMatrix, d_bestMatch, som_dim);

        hipError_t error = hipGetLastError();

        if (error != hipSuccess)
        {
            fprintf(stderr, "Failed to launch CUDA kernel findBestMatchingNeuron_kernel (error code %s)!\n", hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }
    }
    {
		// Setup execution parameters
		int neuron_size = numberOfChannels * neuron_dim * neuron_dim;
		int gridSize = ceil((float)neuron_size/BLOCK_SIZE);
		dim3 dimBlock(BLOCK_SIZE);
		dim3 dimGrid(gridSize, som_dim, som_dim);

		// Start kernel
		updateNeurons_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_bestRotationMatrix,
			d_bestMatch, neuron_size);

		hipError_t error = hipGetLastError();

		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to launch CUDA kernel updateNeurons_kernel (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
    }
}
