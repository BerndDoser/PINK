#include "hip/hip_runtime.h"
/**
 * @file   cuda_calculateEuclideanDistanceWithoutSquareRoot.cu
 * @date   Oct 30, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "CudaLib.h"
#include "hipblas.h"
#include <stdio.h>

#define BLOCK_SIZE 64

/**
 * CUDA Kernel Device code
 *
 * Computes the euclidean distance of two arrays.
 */
template <unsigned int block_size>
__global__ void
euclidean_distance_kernel(float *a, float *b, float *c, int image_size)
{
	int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int som_idx = blockIdx.y;
    int rot_idx = blockIdx.z;

    __shared__ float a_local[block_size];
    __shared__ float b_local[block_size];
    __shared__ float c_local[block_size];

    a_local[tid] = (i < image_size) ? a[i + som_idx * image_size] : 0.0;
    b_local[tid] = (i < image_size) ? b[i + rot_idx * image_size] : 0.0;

    float diff = a_local[tid] - b_local[tid];
    c_local[tid] = diff * diff;
    __syncthreads();

    // parallel reduction
    for (int s=1; s < blockDim.x; s *= 2) {
    	if (tid % (2*s) == 0) {
    	    c_local[tid] += c_local[tid + s];
    	}
    	__syncthreads();
    }

    if (tid == 0) c[blockIdx.x*blockDim.y*blockDim.z + blockIdx.y*blockDim.z + blockIdx.z] = c_local[0];
}

/**
 * CUDA Kernel Device code
 *
 * Computes the euclidean distance of two arrays.
 */
template <unsigned int block_size>
__global__ void
euclidean_distance_kernel2(float *a, float *b, float *c, int image_size)
{
	int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int som_idx = blockIdx.y;
    int rot_idx = blockIdx.z;

	__shared__ float c_local[block_size];

    if (i < image_size) {
		float diff1 = a[i + som_idx * image_size] - b[i + rot_idx * image_size];
		float diff2 = a[i + som_idx * image_size + block_size] - b[i + rot_idx * image_size + block_size];
		c_local[tid] = diff1 * diff1;
		c_local[tid + block_size] = diff2 * diff2;
    } else {
    	c_local[tid] = 0.0f;
    }

	__syncthreads();

	if (tid < 32) {
		if (block_size >= 64) c_local[tid] += c_local[tid + 32];
		if (block_size >= 32) c_local[tid] += c_local[tid + 16];
		if (block_size >= 16) c_local[tid] += c_local[tid +  8];
		if (block_size >=  8) c_local[tid] += c_local[tid +  4];
		if (block_size >=  4) c_local[tid] += c_local[tid +  2];
		if (block_size >=  2) c_local[tid] += c_local[tid +  1];
	}

	if (tid == 0) c[blockIdx.x*blockDim.y*blockDim.z + blockIdx.y*blockDim.z + blockIdx.z] = c_local[0];
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
void cuda_generateEuclideanDistanceMatrix(float *d_euclideanDistanceMatrix, int *d_bestRotationMatrix, int som_dim, float* d_som,
    int image_dim, int num_rot, float* d_rotatedImages)
{
	unsigned int image_size = image_dim * image_dim;
	unsigned int som_size = som_dim * som_dim;
	unsigned int red_size = ((float)image_size/BLOCK_SIZE);

    // Setup execution parameters
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(red_size, som_size, num_rot);

    float *d_tmp = cuda_alloc_float(dimGrid.x * dimGrid.y * dimGrid.z);

    // Start kernel
    //printf("Starting CUDA Kernel with (%i,%i,%i) blocks and (%i,%i,%i) threads ...\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);
    euclidean_distance_kernel<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_som, d_rotatedImages, d_tmp, image_size);

    hipError_t error = hipGetLastError();

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch CUDA kernel (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Call second step for final reduction
    cuda_generateEuclideanDistanceMatrix_secondStep(d_euclideanDistanceMatrix, d_bestRotationMatrix, d_tmp,
        image_size, num_rot, red_size);

    cuda_free(d_tmp);
}
