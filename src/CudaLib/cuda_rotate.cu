#include "hip/hip_runtime.h"
/**
 * @file   cuda_rotate.cu
 * @date   Oct 17, 2014
 * @author Bernd Doser, HITS gGmbH
 */

#include "cuda_rotate.h"
#include <stdio.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
rotate_kernel(const float *source, float *dest, int height, int width, float alpha)
{
    int i  = blockDim.x * blockIdx.x + threadIdx.x;
    int j  = blockDim.y * blockIdx.y + threadIdx.y;
    int is = blockDim.x * gridDim.x;
    int js = blockDim.y * gridDim.y;

    int x0, x1, x2, y0, y1, y2;
    const float cosAlpha = cos(alpha);
    const float sinAlpha = sin(alpha);

    x0 = width / 2;
    y0 = height / 2;

    for (x1 = i; x1 < width; x1 += is) {
        for (y1 = j; y1 < height; y1 += js) {
        	x2 = (x1 - x0) * cosAlpha - (y1 - y0) * sinAlpha + x0;
        	y2 = (x1 - x0) * sinAlpha + (y1 - y0) * cosAlpha + y0;
            if (x2 > -1 && x2 < width && y2 > -1 && y2 < height) dest[x2*height + y2] = source[x1*height + y1];
        }
    }
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
void cuda_rotate(int height, int width, float *source, float *dest, float alpha)
{
    unsigned int size = height * width;
    unsigned int sizeInBytes = size * sizeof(float);

    // Allocate device memory
    float *d_source, *d_dest;

    hipError_t error;

    error = hipMalloc((void **) &d_source, sizeInBytes);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_source returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_dest, sizeInBytes);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_dest returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_source, source, sizeInBytes, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_source, source) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Setup execution parameters
    const unsigned int blockSize = 32;
    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid(height/blockSize, width/blockSize);
    //dim3 dimBlock(1,1);
    //dim3 dimGrid(1,1);

    printf("Starting CUDA Kernel with (%i,%i,%i) blocks and (%i,%i,%i) threads ...\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);

    rotate_kernel<<<dimGrid, dimBlock>>>(d_source, d_dest, height, width, alpha);

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Copy the device result vector in device memory to the host result vector in host memory.
    error = hipMemcpy(dest, d_dest, sizeInBytes, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to copy d_dest to host (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    error = hipFree(d_dest);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_dest (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipFree(d_source);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to free d_source (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}
